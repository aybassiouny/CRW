#include "hip/hip_runtime.h"
#define DYNAMICEDATA 1

#include <string>
//#include <posix4win.h>
#include "graphchi_basic_includes.hpp"
#include "api/dynamicdata/chivector.hpp"
#include "util/toplist.hpp"
#include <utility>
#include <algorithm> 

using namespace graphchi;
using namespace std;
/**
 * Type definitions. Remember to create suitable graph shards using the
 * Sharder-program.
 */

typedef unsigned int VertexDataType;
typedef chivector<vid_t> pvid_t;
typedef pvid_t EdgeDataType;

vector< vector<vid_t> > walks;
 
const int _walks_per_source = 10, 
            _steps_per_walk = 5;


__global__ void update(int *inc, int *outc, int *rpt, int **res )//int ***vData,int *Voutc, int** res)
    //graphchi_vertex<VertexDataType, EdgeDataType > &vertex, graphchi_context &gcontext) 
{
    int curVer = cudastuff;
    for(int i=0; i<rpt[curVer]; i++){
        int *edge_rand =  new int; 
        *edge_rand = rand()%outc[curVer];
        res[curVer][i] = *edge_rand;
        atomicAdd(rpt+(*edge_rand), 1);
    }
    
    // if (iter == 0) {
    //     for(int i=0; i < _walks_per_source; i++) {
    //         int edge_rand = rand()%outc;
    //          //graphchi_edge<EdgeDataType> * outedge = vertex.random_outedge();
    //          vector<vid_t> walk;
    //          chivector<vid_t> * evector = outedge->get_vector();
    //          int x = vertex.id()*walks_per_source()+i;
    //          evector->add(x);
    //          //return outedge->vertex_id(); // Schedule destination
    //     }
    // } else {
    //     for(int i=0; i < inc; i++) {
    //         graphchi_edge<EdgeDataType> * edge = vertex.inedge(i);
    //         chivector<vid_t> *invector = edge->get_vector();
            
    //         for (int j = 0; j < invector->size(); j++){
    //             if (walks[invector->get(j)].size() < steps_per_walk()){
    //                 graphchi_edge<EdgeDataType> * outedge = vertex.random_outedge();
    //                 chivector<vid_t> *outvector = outedge->get_vector();
                    
    //                 walks[invector->get(j)].push_back(vertex.id());
                    
    //                 outvector->add(invector->get(j));
    //                 gcontext.scheduler->add_task(outedge->vertex_id()); // Schedule destination
    //             }
    //         }
    //         invector->clear();
    //     }
        
    // }
    
}


//static int x =0;
pthread_mutex_t lock;
struct RandomWalkProgram : public GraphChiProgram<VertexDataType, EdgeDataType> {

    int steps_per_walk()
    {
        return _steps_per_walk;
    }
    int walks_per_source() 
    {
        return _walks_per_source;
    }
    
    /**
     *  Vertex update function.
     */
    void update(graphchi_vertex<VertexDataType, EdgeDataType > &vertex, graphchi_context &gcontext) {
        
        if (gcontext.iteration == 0) {
            
            for(int i=0; i < walks_per_source(); i++) {
                 graphchi_edge<EdgeDataType> * outedge = vertex.random_outedge();
                 if (outedge != NULL) {
                     vector<vid_t> walk;
                     chivector<vid_t> * evector = outedge->get_vector();
                     int x = vertex.id()*walks_per_source()+i;
                     evector->add(x);
                     //cout<<vertex.id()<<" "<<x<<endl;
                     gcontext.scheduler->add_task(outedge->vertex_id()); // Schedule destination
                     
                 }
                 else cout<<"WTF"<<endl;
            }
        } else {
            for(int i=0; i < vertex.num_inedges(); i++) {
                graphchi_edge<EdgeDataType> * edge = vertex.inedge(i);
                chivector<vid_t> *invector = edge->get_vector();
                
                for (int j = 0; j < invector->size(); j++){
                    if (walks[invector->get(j)].size() < steps_per_walk()){
                        graphchi_edge<EdgeDataType> * outedge = vertex.random_outedge();
                        if (outedge != NULL) {
                            chivector<vid_t> *outvector = outedge->get_vector();
                            
                            walks[invector->get(j)].push_back(vertex.id());
                            
                            outvector->add(invector->get(j));
                            gcontext.scheduler->add_task(outedge->vertex_id()); // Schedule destination
                        }
                        else cout<<"WTF"<<endl;
                    }
                }
                
                invector->clear();
            }
            
        }
    }
    
    /**
     * Called before an iteration starts.
     */
    void before_iteration(int iteration, graphchi_context &gcontext) {
    }
    
    /**
     * Called after an iteration has finished.
     */
    void after_iteration(int iteration, graphchi_context &gcontext) {
        
    }
    
    /**
     * Called before an execution interval is started.
     */
    void before_exec_interval(vid_t window_st, vid_t window_en, graphchi_context &gcontext) {
    }
    
    /**
     * Called after an execution interval has finished.
     */
    void after_exec_interval(vid_t window_st, vid_t window_en, graphchi_context &gcontext) {
        if (gcontext.iteration == 0){
            cout<<"I am after iteration 0"<<endl;
        }
    }
    
};

 

int main(int argc, const char ** argv) {
    /* GraphChi initialization will read the command line
     arguments and the configuration file. */
    graphchi_init(argc, argv);
    
    /* Metrics object for keeping track of performance counters
     and other information. Currently required. */
    metrics m("randomwalk");
    
    /* Basic arguments for application */
    std::string filename = get_option_string("file");  // Base filename
    int numV = get_option_int("numV");  
    int niters           = get_option_int("niters", 100); // Number of iterations
    bool scheduler       = true;                       // Whether to use selective scheduling
    
    /* Detect the number of shards or preprocess an input to create them */
    bool preexisting_shards;
    int nshards          = convert_if_notexists<vid_t>(filename, get_option_string("nshards", "auto"), preexisting_shards);
    
    


    /* Run */
    RandomWalkProgram program;
    
    //initialize walks

    int numWalks= program.walks_per_source()*numV;
    cout<<"numWalks is "<<numWalks<<endl;

    ofstream out("walks.txt");;
    for(int i=0; i<numV; i++){
        vector<vid_t> walk; walk.push_back(i);
        for(int j=0; j<program.walks_per_source(); j++)
            walks.push_back(walk);
    }
    // for(int i=0; i < min(int(walks.size()), 1000); i++) {
    //     for (int j = 0; j < walks[i].size(); j++)
    //         out << walks[i][j]<<" ";
    //     out << std::endl;
    // }
    // out<<"#################################################################"<<endl;
    // out<<"#################################################################"<<endl;
    // out<<"#################################################################"<<endl;

    //go
    graphchi_engine<VertexDataType, EdgeDataType> engine(filename, nshards, scheduler, m);
    if (preexisting_shards) {
        engine.reinitialize_edge_data(0);
    }
    engine.run(program, niters);
    
    /* List top 20 */
    //std::vector< vertex_value<VertexDataType> > top = get_top_vertices<VertexDataType>(filename, ntop);
    //std::cout << "Print top 20 vertices: " << std::endl;
    
    
    for(int i=0; i < min(int(walks.size()), 1000); i++) {
        for (int j = 0; j < walks[i].size(); j++)
            out << walks[i][j]<<" ";
        out << std::endl;
    }
    out.close();
    out<< walks.size()<<" "<<walks[0].size()<<endl;
    /* Report execution metrics */
    metrics_report(m);
    return 0;
}
