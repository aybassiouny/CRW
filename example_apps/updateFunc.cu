#include "hip/hip_runtime.h"
#include "updateFunc.h"

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void setup_kernel(hiprandState *state){

  unsigned int seed = (unsigned int) clock64();
  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void update2(int *inc, int *outc, int *rpt, int *res, int *mxrptV , int *numgoodV, hiprandState *my_curandstate ) 
{
	int curVer = blockIdx.x * blockDim.x + threadIdx.x;
	//for (int curVer = 0; curVer<*numgoodV; curVer++){
		for (int i = 0; i<rpt[curVer]; i++){
			//int *edge_rand = new int;
			//*edge_rand = rand() % outc[curVer];
            float myrandf = hiprand_uniform(my_curandstate+curVer);
            myrandf *= (outc[curVer] - 0+0.999999);
            myrandf += 0;
            int edge_rand = (int)truncf(myrandf);
			res[curVer*(*mxrptV) + i] = edge_rand;
			//atomicAdd(rpt+(edge_rand), 1);
		}
	//}
}


void callUpdate2(int *inc, int *outc, int *minisch, int *res, int mxrptV , int numgoodV )
{
    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));
    setup_kernel<<<1,1>>>(d_state);
    
	int *inc_d, *outc_d, *minisch_d, *res_d, *mxrptV_d, *numgoodV_d;
                
    HANDLE_ERROR( hipMalloc( (void**)&inc_d, numgoodV * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&outc_d, numgoodV * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&minisch_d, numgoodV * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&res_d, numgoodV*mxrptV * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&mxrptV_d, 1 * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&numgoodV_d, 1 * sizeof(int) ) );
    
    HANDLE_ERROR( hipMemcpy( inc_d, inc, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( outc_d, outc, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( minisch_d, minisch, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( res_d, res,  numgoodV*mxrptV * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( mxrptV_d, &mxrptV, 1 * sizeof(int), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( numgoodV_d, &numgoodV, 1 * sizeof(int), hipMemcpyHostToDevice ) );
    
    update2 <<< 500,500 >>> (inc_d,  outc_d,  minisch_d,  res_d,  mxrptV_d, numgoodV_d, d_state);
    
    HANDLE_ERROR( hipMemcpy( inc, inc_d, numgoodV * sizeof(int), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( outc, outc_d, numgoodV * sizeof(int), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( minisch, minisch_d, numgoodV * sizeof(int), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( res, res_d,  numgoodV*mxrptV * sizeof(int), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( &mxrptV, mxrptV_d, 1 * sizeof(int), hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( &numgoodV, numgoodV_d, 1 * sizeof(int), hipMemcpyDeviceToHost ) );
    
	hipFree( inc_d );
	hipFree( outc_d );
	hipFree( minisch_d );
	hipFree( res_d );
	hipFree( mxrptV_d );
	hipFree( numgoodV_d );
}

