#include "hip/hip_runtime.h"
#include "updateFunc.h"

const int CUDA_MAX_BLOCKS =65535;
const int CUDA_MAX_THREADS =1024;
const int CUDA_MAX_V = 100000;

//const int CUDA_MAX_THREADS_PER_BLOCK =65535;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void setup_kernel(hiprandState *state, int *numgoodV){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<*numgoodV){
      unsigned int seed = (unsigned int) clock64()+idx;
      hiprand_init(seed, idx, 0, &state[idx]);
  }
}

__global__ void update2(int *inc, int *outc, int *rpt, int *res, int *resloc, int *numgoodV, int *offset, hiprandState *my_curandstate) 
{
    int curVer = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("Currently doing thread %d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
    if(curVer<*numgoodV)
    {
        for (int i = 0; i<rpt[curVer]; i++){
            
            float myrandf = hiprand_uniform(my_curandstate+curVer);
            myrandf *= (outc[curVer]-1 - 0+0.999999);
            myrandf += 0;
            res[resloc[curVer]-*offset+i] = (int)truncf(myrandf);
        }
    }
}


void callUpdate2(int *inc, int *outc, int *minisch, int *res, int* resloc , int totressz, int totgoodV )
{
    int remgoodV = totgoodV;
    int offset=0;
    int ressz =0;
    //for(int curiter = 0; curiter<=niter; curiter++)
    while(remgoodV>0)
    {
        assert(offset<totgoodV);
        int numgoodV = min(remgoodV, CUDA_MAX_V);
        int resloc_offset = resloc[offset];
        ressz = resloc[numgoodV+offset-1]+minisch[numgoodV+offset-1] - 
            (resloc[offset]);
        std::cout<<numgoodV<<" "<<ressz<<" "<<offset<<" "<<std::endl;
        hiprandState *d_state;
        hipMalloc(&d_state, numgoodV*sizeof(hiprandState));
        int numThreads  = 256;
        int numBlocks = (numgoodV+numThreads-1)/numThreads;  
        
        assert(numThreads<=CUDA_MAX_THREADS);
        assert(numBlocks<=CUDA_MAX_BLOCKS);

        

        int *inc_d, *outc_d, *minisch_d, *res_d, *resloc_d, *numgoodV_d, *offset_d;
                    
        HANDLE_ERROR( hipMalloc( (void**)&inc_d, numgoodV * sizeof(int) ) );
        HANDLE_ERROR( hipMalloc( (void**)&outc_d, numgoodV * sizeof(int) ) );
        HANDLE_ERROR( hipMalloc( (void**)&minisch_d, numgoodV * sizeof(int) ) );
        HANDLE_ERROR( hipMalloc( (void**)&res_d, ressz * sizeof(int) ) );
        HANDLE_ERROR( hipMalloc( (void**)&resloc_d, numgoodV * sizeof(int) ) );
        HANDLE_ERROR( hipMalloc( (void**)&numgoodV_d, 1 * sizeof(int) ) );
        HANDLE_ERROR( hipMalloc( (void**)&offset_d, 1 * sizeof(int) ) );
        
        HANDLE_ERROR( hipMemcpy( inc_d, inc+offset, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( outc_d, outc+offset, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( minisch_d, minisch+offset, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( resloc_d, resloc+offset, numgoodV * sizeof(int), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( res_d, res+resloc[offset],  ressz* sizeof(int), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( numgoodV_d, &numgoodV, 1 * sizeof(int), hipMemcpyHostToDevice ) );
        HANDLE_ERROR( hipMemcpy( offset_d, &resloc_offset, 1 * sizeof(int), hipMemcpyHostToDevice ) );
        
         
        setup_kernel<<<numBlocks,numThreads>>>(d_state, numgoodV_d);
        update2 <<<numBlocks,numThreads>>> (inc_d, outc_d, minisch_d, res_d, resloc_d, numgoodV_d, offset_d, d_state);
        
        HANDLE_ERROR( hipMemcpy( res+resloc[offset], res_d,  ressz * sizeof(int), hipMemcpyDeviceToHost ) );
        
        hipFree( inc_d );
        hipFree(d_state);
        hipFree( outc_d );
        hipFree( minisch_d );
        hipFree( res_d );
        hipFree( resloc_d );
        //hipFree( mxrptV_d );
        hipFree( numgoodV_d );
        hipFree( offset_d );

        remgoodV -= CUDA_MAX_V;
        offset+=CUDA_MAX_V;
    }
}